
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>


#define N 100000
#define K 10
#define MAX_ITER 50
#define TPB 128
#define EPSILON 0.00001

__device__ float distance_2D(const float x1, const float x2, const float y1, const float y2)
{
    return sqrt(pow((x1-y1),2) + pow((x2-y2),2));
}


__global__ void kMeansClusterAssignment(const float *d_datapoints_x, const float *d_datapoints_y, int *d_clust_assn, const float *d_centroids_x, const float *d_centroids_y)
{
    //get idx for this datapoint
    const int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= N) return;


    //find the closest centroid to this datapoint
    float min_dist = INFINITY;
    int closest_centroid = 0;

    for(int c = 0; c<K; ++c)

    {
        float dist = distance_2D(d_datapoints_x[idx], d_datapoints_y[idx], d_centroids_x[c], d_centroids_y[c]);

        if(dist < min_dist)
        {
            min_dist = dist;
            closest_centroid=c;
        }
    }

    //assign closest cluster id for this datapoint/thread
    d_clust_assn[idx]=closest_centroid;
    //d_clust_sizes[closest_centroid]+=1;
}
__global__ void kMeansCentroidUpdate_Sum(const float *d_datapoints_x, const float *d_datapoints_y, const int *d_clust_assn, float *d_centroids_sum_x, float *d_centroids_sum_y, int *d_clust_sizes) {

    //get idx of thread at grid level
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N) return;

    int clust_id = d_clust_assn[idx];

    atomicAdd(&(d_centroids_sum_x[clust_id]), d_datapoints_x[idx]);
    atomicAdd(&(d_centroids_sum_y[clust_id]), d_datapoints_y[idx]);
    atomicAdd(&(d_clust_sizes[clust_id]), 1);
    //d_clust_sizes[clust_id]+=1;


}

//Function to stop the algorithm when convergence is reached
int compareArrays(float a[], float b[], int n) {
    int i;
    for(i=0; i<n; i++){
        if(abs(a[i]-b[i]) > EPSILON)
            return 0;
    }
    return 1;
}


int main()
{
    srand(time(NULL));   // Initialization, should only be called once.
    FILE *fpt;
    //FILE *fpt_centroids;

    //fpt = fopen("/home/federico/CLionProjects/kmeans_cuda/datasets/2D_data_3.csv", "r");
    fpt = fopen("/home/federico/CLionProjects/kmeans_cuda/datasets/2D_data_uniform.csv", "r");
    //fpt_centroids = fopen("/home/federico/CLionProjects/kmeans_cuda/datasets/2D_data_3_centroids.csv", "r");

    //allocate memory on the device for the data points
    float *d_datapoints_x;
    float *d_datapoints_y;
    //allocate memory on the device for the cluster assignments
    int *d_clust_assn;
    //allocate memory on the device for the cluster centroids
    float *d_centroids_sum_x;
    float *d_centroids_sum_y;
    float *d_centroids_x;
    float *d_centroids_y;
    //allocate memory on the device for the cluster sizes
    int *d_clust_sizes;

    hipMalloc(&d_datapoints_x, N*sizeof(float));
    hipMalloc(&d_datapoints_y, N*sizeof(float));
    hipMalloc(&d_clust_assn,N*sizeof(int));
    hipMalloc(&d_centroids_sum_x,K*sizeof(float));
    hipMalloc(&d_centroids_sum_y,K*sizeof(float));
    hipMalloc(&d_centroids_x,K*sizeof(float));
    hipMalloc(&d_centroids_y,K*sizeof(float));
    hipMalloc(&d_clust_sizes,K*sizeof(int));

    //allocate memory for host
    float *h_centroids_x = (float*)malloc(K*sizeof(float));
    float *h_centroids_y = (float*)malloc(K*sizeof(float));
    float *h_centroids_sum_x = (float*)malloc(K*sizeof(float));
    float *h_centroids_sum_y = (float*)malloc(K*sizeof(float));
    float *h_datapoints_x = (float*)malloc(N*sizeof(float));
    float *h_datapoints_y = (float*)malloc(N*sizeof(float));
    int *h_clust_assn = (int*)malloc(N*sizeof(int));
    int *h_clust_sizes = (int*)malloc(K*sizeof(int));



    //initalize datapoints from csv
    printf("DataPoints: \n");
    for(int i=0;i<N;++i){
        fscanf(fpt,"%f,%f\n", &h_datapoints_x[i], &h_datapoints_y[i]);
        printf("(%f, %f) \n",  h_datapoints_x[i], h_datapoints_y[i]);
    }
    fclose(fpt);


    //initialize centroids, choose k-random points from datapoints
    printf("Clusters: \n");
    for(int i=0;i<K;++i){
        int r = rand() % N;
        h_centroids_x[i] = h_datapoints_x[r];
        h_centroids_y[i] = h_datapoints_y[r];
        h_centroids_sum_x[i]=0.0;
        h_centroids_sum_y[i]=0.0;
        printf("(%f, %f) \n",  h_centroids_x[i], h_centroids_y[i]);
        h_clust_sizes[i]=0;
    }



    //copy datapoints and all other data from host to device
    hipMemcpy(d_centroids_x,h_centroids_x,K*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_centroids_y,h_centroids_y,K*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_centroids_sum_x,h_centroids_sum_x,K*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_centroids_sum_y,h_centroids_sum_y,K*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_datapoints_x,h_datapoints_x,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_datapoints_y,h_datapoints_y,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_clust_sizes,h_clust_sizes,K*sizeof(int),hipMemcpyHostToDevice);



    //Start time for clustering
    clock_t start = clock();
    int cur_iter = 0;

    while(cur_iter < MAX_ITER)
    {
        printf("Iter %d: \n",cur_iter);
        //Start time for iteration
        clock_t start_iter = clock();


        //Points assg
        kMeansClusterAssignment<<<(N+TPB-1)/TPB, TPB>>>(d_datapoints_x, d_datapoints_y, d_clust_assn, d_centroids_x, d_centroids_y);

        //cudaMemcpy(h_clust_sizes,d_clust_sizes,K*sizeof(int),cudaMemcpyDeviceToHost);


        //reset centroids and cluster sizes (will be updated in the next kernel)
        hipMemset(d_centroids_sum_x,0.0,K*sizeof(float));
        hipMemset(d_centroids_sum_y,0.0,K*sizeof(float));

        //call centroid update
        kMeansCentroidUpdate_Sum<<<(N+TPB-1)/TPB, TPB>>>(d_datapoints_x, d_datapoints_y, d_clust_assn, d_centroids_sum_x, d_centroids_sum_y, d_clust_sizes);

        //Copy centroids sum and clusters sizes back to host
        hipMemcpy(h_centroids_sum_x,d_centroids_sum_x,K*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(h_centroids_sum_y,d_centroids_sum_y,K*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(h_clust_sizes,d_clust_sizes,K*sizeof(int),hipMemcpyDeviceToHost);

        hipMemset(d_clust_sizes,0,K*sizeof(int));
        for(int i=0; i < K; i++){
            h_centroids_x[i]=h_centroids_sum_x[i]/h_clust_sizes[i];
            h_centroids_y[i]=h_centroids_sum_y[i]/h_clust_sizes[i];
        }
        for(int i=0; i < K; i++){
            printf("C %d: (%f, %f)\n",i,h_centroids_x[i],h_centroids_y[i]);
        }

        //Stop time for iteration
        clock_t end_iter = clock();
        float seconds_iter = (float)(end_iter - start_iter) / CLOCKS_PER_SEC;
        printf("Time for iter: %f\n", seconds_iter);

        //Compare the centroids for stop the clustering
        hipMemcpy(d_centroids_x,h_centroids_x,K*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(d_centroids_y,h_centroids_y,K*sizeof(float),hipMemcpyHostToDevice);

        cur_iter+=1;
    }

    hipMemcpy(h_clust_assn,d_clust_assn,N*sizeof(int),hipMemcpyDeviceToHost);

    clock_t end = clock();
    float seconds = (float)(end - start) / CLOCKS_PER_SEC;
    printf("Time for clustering: %f\n", seconds);

    FILE *res;

    res = fopen("/home/federico/CLionProjects/kmeans_cuda/results/2D_data_3_results.csv", "w+");
    for(int i=0;i<N;i++){
        fprintf(res,"%d\n", h_clust_assn[i]);
    }

    hipFree(d_datapoints_x);
    hipFree(d_datapoints_y);
    hipFree(d_clust_assn);
    hipFree(d_centroids_x);
    hipFree(d_centroids_y);
    hipFree(d_clust_sizes);

    free(h_centroids_x);
    free(h_centroids_y);
    free(h_datapoints_x);
    free(h_datapoints_y);
    free(h_clust_assn);
    free(h_clust_sizes);

    return 0;
}


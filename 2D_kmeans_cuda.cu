
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>


#define N 100000
#define K 4
#define MAX_ITER 50
#define TPB 256
#define EPSILON 0.00000001

__device__ float distance_2D(const float x1, const float x2, const float y1, const float y2)
{
    return sqrt(pow((x1-y1),2) + pow((x2-y2),2));
}


__global__ void kMeansClusterAssignment(const float *d_datapoints_x, const float *d_datapoints_y, int *d_clust_assn, const float *d_centroids_x, const float *d_centroids_y)
{
    //get idx for this datapoint
    const int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= N) return;


    //find the closest centroid to this datapoint
    float min_dist = INFINITY;
    int closest_centroid = 0;

    for(int c = 0; c<K; ++c)

    {
        float dist = distance_2D(d_datapoints_x[idx], d_datapoints_y[idx], d_centroids_x[c], d_centroids_y[c]);

        if(dist < min_dist)
        {
            min_dist = dist;
            closest_centroid=c;
        }
    }

    //assign closest cluster id for this datapoint/thread
    d_clust_assn[idx]=closest_centroid;
}


__global__ void kMeansCentroidUpdate_Sum(const float *d_datapoints_x, const float *d_datapoints_y, const int *d_clust_assn, float *d_centroids_x, float *d_centroids_y, float *d_clust_sizes)
{

    //get idx of thread at grid level
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;


    if (idx >= N) return;

    //get idx of thread at the block level
    const int s_idx = threadIdx.x;

    //put the datapoints and corresponding cluster assignments in shared memory so that they can be summed by thread 0 later
    __shared__ float s_datapoints_x[TPB];
    s_datapoints_x[s_idx]= d_datapoints_x[idx];

    __shared__ float s_datapoints_y[TPB];
    s_datapoints_y[s_idx]= d_datapoints_y[idx];

    __shared__ int s_clust_assn[TPB];
    s_clust_assn[s_idx] = d_clust_assn[idx];

    __syncthreads();

    //it is the thread with idx 0 (in each block) that sums up all the values within the shared array for the block it is in
    if(s_idx==0)
    {
        float b_clust_datapoint_sums_x[K]={0};
        float b_clust_datapoint_sums_y[K]={0};
        float b_clust_sizes[K]={0};

        for(int j=0; j < blockDim.x; ++j)
        {
            int clust_id = s_clust_assn[j];
            b_clust_datapoint_sums_x[clust_id]+=s_datapoints_x[j];
            b_clust_datapoint_sums_y[clust_id]+=s_datapoints_y[j];
            b_clust_sizes[clust_id]+=1;
        }
        //Now we add the sums to the global centroids and add the counts to the global counts.
        for(int z=0; z < K; ++z)
        {
            atomicAdd(&d_centroids_x[z],b_clust_datapoint_sums_x[z]);
            atomicAdd(&d_centroids_y[z],b_clust_datapoint_sums_y[z]);
            atomicAdd(&d_clust_sizes[z],b_clust_sizes[z]);
        }
    }

    __syncthreads();

    //currently centroids are just sums, so divide by size to get actual centroids
    if(idx < K){
        d_centroids_x[idx] = d_centroids_x[idx]/d_clust_sizes[idx];
        d_centroids_y[idx] = d_centroids_y[idx]/d_clust_sizes[idx];
    }

}
__global__ void kMeansCentroidUpdate_Div(float *d_centroids_x, float *d_centroids_y, float *d_clust_sizes){
    int idx = threadIdx.x;
    d_centroids_x[idx] = d_centroids_x[idx]/d_clust_sizes[idx];
    d_centroids_y[idx] = d_centroids_y[idx]/d_clust_sizes[idx];
}

//Function to stop the algorithm when convergence is reached
int compareArrays(float a[], float b[], int n) {
    int i;
    for(i=0; i<n; i++){
        if(abs(a[i]-b[i]) > EPSILON)
            return 0;
    }
    return 1;
}


int main()
{
    srand(time(NULL));   // Initialization, should only be called once.
    FILE *fpt;
    //FILE *fpt_centroids;

    fpt = fopen("/home/federico/CLionProjects/kmeans_cuda/datasets/2D_data_3.csv", "r");
    //fpt = fopen("/home/federico/CLionProjects/kmeans_cuda/datasets/2D_data_uniform.csv", "r");
    //fpt_centroids = fopen("/home/federico/CLionProjects/kmeans_cuda/datasets/2D_data_3_centroids.csv", "r");

    //allocate memory on the device for the data points
    float *d_datapoints_x;
    float *d_datapoints_y;
    //allocate memory on the device for the cluster assignments
    int *d_clust_assn;
    //allocate memory on the device for the cluster centroids
    float *d_centroids_x;
    float *d_centroids_y;
    //allocate memory on the device for the cluster sizes
    float *d_clust_sizes;

    hipMalloc(&d_datapoints_x, N*sizeof(float));
    hipMalloc(&d_datapoints_y, N*sizeof(float));
    hipMalloc(&d_clust_assn,N*sizeof(int));
    hipMalloc(&d_centroids_x,K*sizeof(float));
    hipMalloc(&d_centroids_y,K*sizeof(float));
    hipMalloc(&d_clust_sizes,K*sizeof(float));

    //allocate memory for host
    float *h_centroids_x = (float*)malloc(K*sizeof(float));
    float *h_centroids_y = (float*)malloc(K*sizeof(float));
    float *h_datapoints_x = (float*)malloc(N*sizeof(float));
    float *h_datapoints_y = (float*)malloc(N*sizeof(float));
    float *h_current_centroids_x = (float*)malloc(K*sizeof(float));
    float *h_current_centroids_y = (float*)malloc(K*sizeof(float));
    int *h_clust_assn = (int*)malloc(N*sizeof(int));
    int *h_clust_sizes = (int*)malloc(K*sizeof(float));



    //initalize datapoints from csv
    printf("DataPoints: \n");
    for(int i=0;i<N;++i){
        fscanf(fpt,"%f,%f\n", &h_datapoints_x[i], &h_datapoints_y[i]);
        printf("(%f, %f) \n",  h_datapoints_x[i], h_datapoints_y[i]);
    }
    fclose(fpt);


    //initialize centroids, choose k-random points from datapoints
    printf("Clusters: \n");
    for(int i=0;i<K;++i){
        int r = rand() % N;
        h_centroids_x[i] = h_datapoints_x[r];
        h_centroids_y[i] = h_datapoints_y[r];
        h_current_centroids_x[i]=0.0;
        h_current_centroids_y[i]=0.0;
        printf("(%f, %f) \n",  h_centroids_x[i], h_centroids_y[i]);
        h_clust_sizes[i]=0;
    }



    //copy datapoints and all other data from host to device
    hipMemcpy(d_centroids_x,h_centroids_x,K*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_centroids_y,h_centroids_y,K*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_datapoints_x,h_datapoints_x,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_datapoints_y,h_datapoints_y,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_clust_sizes,h_clust_sizes,K*sizeof(float),hipMemcpyHostToDevice);



    //Start time for clustering
    clock_t start = clock();
    int cur_iter = 0;

    while(cur_iter < MAX_ITER)
    {
        printf("Iter %d: \n",cur_iter);
        //Start time for iteration
        clock_t start_iter = clock();

        //Points assg
        kMeansClusterAssignment<<<(N+TPB-1)/TPB, TPB>>>(d_datapoints_x, d_datapoints_y, d_clust_assn, d_centroids_x, d_centroids_y);

        //copy new centroids back to host
        hipMemcpy(h_centroids_x,d_centroids_x,K*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(h_centroids_y,d_centroids_y,K*sizeof(float),hipMemcpyDeviceToHost);

        //copy assg back to host
        hipMemcpy(h_clust_assn,d_clust_assn,N*sizeof(int),hipMemcpyDeviceToHost);

        for(int i =0; i < K; i++){
            printf("C %d: (%f, %f)\n",i,h_centroids_x[i],h_centroids_y[i]);
        }
        //reset centroids and cluster sizes (will be updated in the next kernel)
        hipMemset(d_centroids_x,0.0,K*sizeof(float));
        hipMemset(d_centroids_y,0.0,K*sizeof(float));
        hipMemset(d_clust_sizes,0,K*sizeof(int));

        //call centroid update
        kMeansCentroidUpdate_Sum<<<(N+TPB-1)/TPB, TPB>>>(d_datapoints_x, d_datapoints_y, d_clust_assn, d_centroids_x, d_centroids_y, d_clust_sizes);

        //kMeansCentroidUpdate_Div<<<1, K>>>(d_centroids_x, d_centroids_y, d_clust_sizes);

/*
        cudaMemcpy(h_current_centroids_x,d_centroids_x,K*sizeof(float),cudaMemcpyDeviceToHost);
        cudaMemcpy(h_current_centroids_y,d_centroids_y,K*sizeof(float),cudaMemcpyDeviceToHost);
        for(int i=0;i<K;++i){
            printf("h(%f, %f) \n",  h_current_centroids_x[i], h_current_centroids_y[i]);
            h_clust_sizes[i]=0;
        }
*/


        //Stop time for iteration
        clock_t end_iter = clock();
        float seconds_iter = (float)(end_iter - start_iter) / CLOCKS_PER_SEC;
        printf("Time for iter: %f\n", seconds_iter);

        //Compare the centroids for stop the clustering
        hipMemcpy(h_current_centroids_x,d_centroids_x,K*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(h_current_centroids_y,d_centroids_y,K*sizeof(float),hipMemcpyDeviceToHost);

        if(compareArrays(h_current_centroids_x, h_centroids_x, K) && compareArrays(h_current_centroids_y, h_centroids_y, K))
            break;


        cur_iter+=1;
    }


    clock_t end = clock();
    float seconds = (float)(end - start) / CLOCKS_PER_SEC;
    printf("Time for clustering: %f\n", seconds);

    FILE *res;

    res = fopen("/home/federico/CLionProjects/kmeans_cuda/results/2D_data_3_results.csv", "w+");
    for(int i=0;i<N;i++){
        fprintf(res,"%d\n", h_clust_assn[i]);
    }

    hipFree(d_datapoints_x);
    hipFree(d_datapoints_y);
    hipFree(d_clust_assn);
    hipFree(d_centroids_x);
    hipFree(d_centroids_y);
    hipFree(d_clust_sizes);

    free(h_centroids_x);
    free(h_centroids_y);
    free(h_current_centroids_x);
    free(h_current_centroids_y);
    free(h_datapoints_x);
    free(h_datapoints_y);
    free(h_clust_assn);
    free(h_clust_sizes);

    return 0;
}

